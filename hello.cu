/*
sudo apt install cuda-nvidia-toolkit
nvcc hello.cu -o hello
*/

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void helloCUDA()
{
    printf("Hello CUDA!\n");
}

int main()
{
    // Launch the kernel
    helloCUDA<<<1,1>>>();
    hipDeviceSynchronize();

    // Print a message to indicate completion
    std::cout << "Kernel executed successfully!\n";

    return 0;
}

