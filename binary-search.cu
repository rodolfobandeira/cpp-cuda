#include <iostream>
#include <hip/hip_runtime.h>

#define N 1000000

__global__ void binary_search(int* array, int key, int* result)
{
    int left = 0;
    int right = N - 1;

    while (left <= right) {
        int mid = left + (right - left) / 2;
        if (array[mid] == key) {
            *result = mid;
            return;
        } else if (array[mid] < key) {
            left = mid + 1;
        } else {
            right = mid - 1;
        }
    }

    *result = -1;
}

int main()
{
    int array[N];
    for (int i = 0; i < N; i++) {
        array[i] = i;
    }

    int key = 1234;
    int result = -1;

    int* d_array;
    int* d_result;

    hipMalloc((void**)&d_array, N * sizeof(int));
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_array, array, N * sizeof(int), hipMemcpyHostToDevice);

    binary_search<<<1,1>>>(d_array, key, d_result);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_array);
    hipFree(d_result);

    if (result == -1) {
        std::cout << "Element not found" << std::endl;
    } else {
        std::cout << "Element found at index " << result << std::endl;
    }

    return 0;
}

